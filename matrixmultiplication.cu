// Multiply square matrices on GPU with CUDA and return the product


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <cassert>
using namespace std;

// Pass three matrices ind and the dimensions

__global__ void matrixMul(int *A, int *B, int *C, int n)
// find out which block is being calculated and iterate through each element
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // Iterate through rows and colums
    C[i*n+j] = 0;
    for (int k = 0; k < n; k++)
    {
        C[i*n+j] += A[i*n+k] * B[k*n+j];
    }
}

// Initialize a square matrix of size n with random values between 0 and 100
void init_matrix(int *matrix, int n) {
    int i, j;
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            matrix[i * n + j] = rand() % 100;
        }
    }
}

int main (){
int dimensions = 1 << 10;
// Allocate memory for "dimensions elements"
size_t bytes = dimensions * dimensions * sizeof(int);
// Allocate memory for matrices
int *mat_a, *mat_b, *output;
hipMallocManaged(&mat_a, bytes);
hipMallocManaged(&mat_b, bytes);
hipMallocManaged(&output, bytes);
//Initialize matrices
init_matrix(mat_a, dimensions);
init_matrix(mat_b, dimensions);
// Allocate threads for output
int threads = 16;
int blocks = (threads * threads) / dimensions;
// Kernel parameters
dim3 THREADS (threads, threads);
dim3 BLOCKS (blocks, blocks);

// Launch the kernel
matrixMul <<<BLOCKS,THREADS>>>(mat_a, mat_b, output, dimensions);
hipDeviceSynchronize();
cout<<"Matrix multiplication completed"<<endl;
  hipFree(mat_a);
  hipFree(mat_b);
  hipFree(output);
return 0;
}
